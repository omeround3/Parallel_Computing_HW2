#include "hip/hip_runtime.h"


#include <hip/hip_runtime.h>
#include "cuda_api.h"
#include <stdio.h>


#define MAXIMUM_THREADS 256

__global__ void histogram_calculator(int* arr, int* results, int size)
{
	extern __shared__ int hist_list[];
	int number;
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if(index < size)
	{
		number = arr[index];
		atomicAdd(&(hist_list[number]), 1);
	}
	__syncthreads();

	atomicAdd(&(results[threadIdx.x]), hist_list[threadIdx.x]);
}

/* Free memory from GPU */
void free_memory(int* arr, int* hist_list)
{
	hipFree(hist_list);
	hipFree(arr);
}

/* Get the number of blocks per grid */
int get_blocks_number(int block_size)
{
	int reminder, blocks, sum_of_blocks;

	blocks = block_size / MAXIMUM_THREADS;

	if (block_size % MAXIMUM_THREADS != 0)
		reminder = 1;
	else
		reminder = 0;
		
	sum_of_blocks = blocks + reminder;

	return sum_of_blocks;
}




int* calc_histogram_on_gpu(int* arr, int size)
{
	
	hipError_t cuda_status;
	bool error_flag = false;
	char* error_msg = "";
	int* histogram = (int*)calloc(MAXIMUM_THREADS, sizeof(int));
	int blocks_num;
	int* arr_int = 0;
	int* hist_cuda = 0;
	
	/* Allocation to CUDA Device */
	cuda_status = hipMalloc((void**)&arr_int, sizeof(int) * size);
	if (cuda_status != hipSuccess )
	{
		error_flag = true;
		error_msg = "[ERROR] cuda Malloc";
	}
	
	cuda_status = hipMalloc((void**)&hist_cuda, sizeof(int) * MAXIMUM_THREADS);
	if (cuda_status != hipSuccess && !error_flag)
	{
		error_flag = true;
		error_msg = "[ERROR] cuda Malloc";
	}
	
	/* Copy from Host memory to CUDA Device memory */
	cuda_status = hipMemcpy(arr_int, arr, sizeof(int) * size, hipMemcpyHostToDevice);
	if (cuda_status != hipSuccess && !error_flag)
	{
		error_flag = true;
		error_msg = "[ERROR] cuda MemcpyHostToDevice";
	}
	
	
	cuda_status = hipMemcpy(hist_cuda, histogram, sizeof(int) * MAXIMUM_THREADS, hipMemcpyHostToDevice);
	if (cuda_status != hipSuccess && !error_flag)
	{
		error_flag = true;
		error_msg = "[ERROR] cuda MemcpyHostToDevice";
	}
	
	blocks_num = get_blocks_number(size);
	histogram_calculator<<<blocks_num, MAXIMUM_THREADS, sizeof(int) * MAXIMUM_THREADS>>>(arr_int, hist_cuda, size);
	
	cuda_status = hipDeviceSynchronize();
	if (cuda_status != hipSuccess && !error_flag)
	{
		error_flag = true;
		error_msg = "[ERROR] cuda DeviceSynchronize";
	}
	
	cuda_status = hipMemcpy(histogram, hist_cuda, sizeof(int) * MAXIMUM_THREADS, hipMemcpyDeviceToHost);
	if (cuda_status !=  hipSuccess && !error_flag)
	{
		error_flag = true;
		error_msg = "[ERROR] cuda MemcpyDeviceToHost";
	}

	if (error_flag == true) {
		printf("%s \n", error_msg);
		free_memory(arr_int, hist_cuda);
		return NULL;
	}

	free_memory(arr_int, hist_cuda);
	return histogram;
}
